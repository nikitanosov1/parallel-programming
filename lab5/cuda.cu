
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <iostream>
#include <ctime>
#include "hipblas.h"

using namespace std;

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on GPU
void GPU_fill_rand(hipDoubleComplex* A, int nr_rows_A, int nr_cols_A) {
    // Create a pseudo-random number generator
    hiprandGenerator_t prng;
    hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);
    // Set the seed for the random number generator using the system clock
    hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

    // Fill the array with random numbers on the device
    hiprandGenerateUniformDouble(prng, (double*)A, nr_rows_A * nr_cols_A * 2);
}

/// Multiply the arrays A and B on GPU and save the result in C
// C(m,n) = A(m,k) * B(k,n)
void gpu_blas_mmul(const hipDoubleComplex* A, const hipDoubleComplex* B, hipDoubleComplex* C, const int m, const int k, const int n) {
    int lda = m, ldb = k, ldc = m;
    const hipDoubleComplex alf = make_hipDoubleComplex(1.0, 0.0);
    const hipDoubleComplex bet = make_hipDoubleComplex(0.0, 0.0);
    const hipDoubleComplex* alpha = &alf;
    const hipDoubleComplex* beta = &bet;

    // CUBLAS is starting here
    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    // Do the actual multiplication
    hipblasZgemm(handle,
                HIPBLAS_OP_N,
                HIPBLAS_OP_T,
                m,
                n,
                k,
                alpha,
                A,
                lda,
                B,
                ldb,
                beta,
                C,
                ldc);
    // Destroy the handle
    hipblasDestroy(handle);
}

void print_matrix(hipDoubleComplex* matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            printf("%f + %fi ", matrix[i + j * rows].x, matrix[i + j * rows].y);
        }
        printf("\n");
    }
    printf("\n");
}

void fill_matrix(hipDoubleComplex* matrix, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            matrix[i + j * rows] = make_hipDoubleComplex((rand() + 0.0) / RAND_MAX, (rand() + 0.0) / RAND_MAX);
        }
    }
}

int main() {
    int nr_rows_A, nr_cols_A, nr_rows_B, nr_cols_B, nr_rows_C, nr_cols_C;
    nr_rows_A = nr_cols_A = nr_rows_B = nr_cols_B = nr_rows_C = nr_cols_C = 1250; // 1250, 2500, 5000
    hipDoubleComplex* h_A = (hipDoubleComplex*)malloc(nr_rows_A * nr_cols_A * sizeof(hipDoubleComplex));
    hipDoubleComplex* h_B = (hipDoubleComplex*)malloc(nr_rows_B * nr_cols_B * sizeof(hipDoubleComplex));
    hipDoubleComplex* h_C = (hipDoubleComplex*)malloc(nr_rows_C * nr_cols_C * sizeof(hipDoubleComplex));

    // create pointers into the GPU
    hipDoubleComplex* d_A;
    hipDoubleComplex* d_B;
    hipDoubleComplex* d_C;

    // allocate memory in the GPU
    hipMalloc(&d_A, nr_rows_A * nr_cols_A * sizeof(hipDoubleComplex));
    hipMalloc(&d_B, nr_rows_B * nr_cols_B * sizeof(hipDoubleComplex));
    hipMalloc(&d_C, nr_rows_C * nr_cols_C * sizeof(hipDoubleComplex));

    double start_ = clock();
    
    // GPU_fill_rand(d_A, nr_rows_A, nr_cols_A);
    // GPU_fill_rand(d_B, nr_rows_B, nr_cols_B);
    
    fill_matrix(h_A, nr_rows_A, nr_cols_A);
    fill_matrix(h_B, nr_rows_B, nr_cols_B);
    // copy the vectors into the GPU
    hipMemcpy(d_A, h_A, nr_rows_A * nr_cols_A * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nr_rows_B * nr_cols_B * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    gpu_blas_mmul(d_A, d_B, d_C, nr_rows_A, nr_cols_A, nr_cols_B);

    // cudaMemcpy(h_A, d_A, nr_rows_A * nr_cols_A * sizeof(cuDoubleComplex), cudaMemcpyDeviceToHost);
    // cudaMemcpy(h_B, d_B, nr_rows_B * nr_cols_B * sizeof(cuDoubleComplex), cudaMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C, nr_rows_C * nr_cols_C * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    double end_ = clock();

    printf("C = \n");
    print_matrix(h_C, 5, 5);
    printf("Parallel mul time =  %.6f ", (end_ - start_) / CLOCKS_PER_SEC);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}