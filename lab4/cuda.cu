#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>

// Код функции ядра
// Здесь объяснение почему код ядра именно такой: 
// https://stackoverflow.com/questions/16619274/cuda-griddim-and-blockdim/29076010#29076010
__global__ void addKernel(double* c, double* a, double* b, unsigned int size) {
    // blockDim – число нитей по x, y, z в блоке
    // gridDim – число блоков по x, y, z в сетке
    // Размер сетки по x
    int gridSize = blockDim.x * gridDim.x;
    int start = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = start; i < size; i += gridSize) {
        c[i] = a[i] + b[i];
    }
}


int main(int argc, char* argv[]) {
    printf("\n(GridDim, BlockDim) = (%d, %d)\n", GRID_SIZE, BLOCK_SIZE);
    printf("N = %d\n", NMAX);

    int n2b = NMAX * sizeof(double);
    // Выделение памяти на хосте
    double* a = (double*)calloc(NMAX, sizeof(double));
    double* b = (double*)calloc(NMAX, sizeof(double));
    double* c = (double*)calloc(NMAX, sizeof(double));

    // Инициализация массивов
    for (int i = 0; i < NMAX; i++) {
        a[i] = rand();
        b[i] = rand();
    }


    // Выделение памяти на устройстве
    double* adev = NULL;
    hipError_t cuerr = hipMalloc((void**)&adev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for a: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    double* bdev = NULL;
    cuerr = hipMalloc((void**)&bdev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for b: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    double* cdev = NULL;
    cuerr = hipMalloc((void**)&cdev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for c: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // Создание обработчиков событий
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    cuerr = hipEventCreate(&start);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot create CUDA start event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    cuerr = hipEventCreate(&stop);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot create CUDA end event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // Копирование данных с хоста на девайс
    cuerr = hipMemcpy(adev, a, n2b, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy a array from host to device: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    cuerr = hipMemcpy(bdev, b, n2b, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy b array from host to device: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // Установка точки старта
    cuerr = hipEventRecord(start, 0);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot record CUDA event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    for (int i = 0; i < ITERATIONS; ++i) {

        //Запуск ядра
        addKernel <<< GRID_SIZE, BLOCK_SIZE >>> (cdev, adev, bdev, NMAX);

        cuerr = hipGetLastError();
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot launch CUDA kernel: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }

        // Синхронизация устройств
        cuerr = hipDeviceSynchronize();
        if (cuerr != hipSuccess)
        {
            fprintf(stderr, "Cannot synchronize CUDA kernel: %s\n",
                hipGetErrorString(cuerr));
            return 0;
        }
    }

    // Установка точки окончания
    cuerr = hipEventRecord(stop, 0);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy c array from device to host: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // Копирование результата на хост
    cuerr = hipMemcpy(c, cdev, n2b, hipMemcpyDeviceToHost);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy c array from device to host: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // Расчет времени
    cuerr = hipEventElapsedTime(&gpuTime, start, stop);
    printf("\nCUDA with (GridDim, BlockDim) = (%d, %d). N = %d\n", GRID_SIZE, BLOCK_SIZE, NMAX);
    printf("CUDA TIME OF WORK IS: %.9f\n", gpuTime / 1000 / ITERATIONS);

    // Очищение памяти
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);
    free(a);
    free(b);
    free(c);


    return 0;
}
